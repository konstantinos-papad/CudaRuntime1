﻿
#include "hip/hip_runtime.h"

#include <cstring>
#include <stdio.h>
#include <string>


__global__ void checkPointer(const int* c, const size_t pitch, const size_t num, const size_t nrows, const size_t ncols);

__global__ void checkPointer(const int *c, const size_t pitch, const size_t num, const size_t nrows, const size_t ncols)
{
    const int icol = blockIdx.x * blockDim.x + threadIdx.x;
    const int irow = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = 0;
    int local = 0;
    if (icol < ncols && irow < nrows)
    {
        idx = irow * (pitch / sizeof(int)) + icol;
        local = c[idx];
    }
}

int main()
{
    const size_t nrows = 5;
    const size_t ncols = 15;

    hipError_t allocError;
    hipError_t cpyError;
    hipError_t status;

    int BLOCKX = 16;
    int BLOCKY = 16;

    int* a = new int[nrows * ncols];
    int b[5 * 15];

    for (auto irow = 0; irow < nrows; irow++) 
    {
        for (auto icol = 0; icol < ncols; icol++)
        {
            int idx = irow * ncols + icol;
            a[idx] = idx;
            b[idx] = idx;
        }
    }

    status = hipSetDevice(0);

    int* a_h;
    allocError = hipHostMalloc((void**)&a_h, nrows * ncols * sizeof(int), hipHostMallocDefault);
    memcpy(a_h, a, nrows * ncols * sizeof(int));
    for (auto i = 0; i < nrows * ncols; i++) 
    {
        printf("%d\n", a_h[i]);
    }

    int* a_d;
    size_t pitch = 0;
    allocError = hipMallocPitch((void**)&a_d, &pitch, ncols * sizeof(int), nrows);
    cpyError = hipMemcpy2D((void*)a_d, pitch, (void*)a_h, ncols * sizeof(int), ncols * sizeof(int), nrows, hipMemcpyHostToDevice);

    dim3 block(BLOCKX, BLOCKY, 1);
    dim3 grid((ncols + block.x - 1) / block.x, (nrows + block.y - 1) / block.y, 1);

    checkPointer<<<block, grid>>>(a_d, pitch, nrows * ncols, nrows, ncols);
    
    status = hipGetLastError();

    status = hipDeviceSynchronize();

    status = hipDeviceReset();
    
    hipHostFree(a_h);
    hipFree(a_d);
    delete[] a;

    return 0;
}